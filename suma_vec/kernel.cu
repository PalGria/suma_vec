
#include "hip/hip_runtime.h"

#include <iostream>

__global__ void addKernel(int *c, const int *a, const int *b)
{

}

void suma_vectores_no_CUDA
(
	float *pA,
	float *pB,
	float *pC,
	const int & crNumElements
)
{
	for (int i = 0; i < crNumElements; ++i)
	{
		pC[i] = pA[i] + pB[i];
	}
}
__global__

void kernel_suma_vectores
(
	const float* cpA,
	const float* cpB,
	float* pC,
	const int cNumElements
)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	//compruebo que tengo hilos suficientes para todos los elementos y por tanto no me salgo
	if(idx < cNumElements)
		pC[idx] = cpA[idx] + cpB[idx];
}

__global__
void suma_vectores_grande(
	const float *cpA,
	const float *cpB,
	float *pC,
	const int cNumElements)
{
	int i;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	for (i = threadIdx.x + blockIdx.x * blockDim.x; i < cNumElements; i += gridDim.x * blockDim.x) {
		pC[i] = cpA[i] + cpB[i];
	}
}

int main()
{
    //paso 1 -> Inicialización
	hipSetDevice(0); //Esta función le dice al framework: voy a usar la tarjeta x

	//paso 2 -> Declaración y reserva
	const int kNumElements = 25600; //numero al azar NO TAN AL AZAR D:<
	size_t kNumBytes = kNumElements * sizeof(float); //bytes totales para reservar y pasarselo al malloc
	//declaro los vectores en la CPU (HOST)
	float *h_A_ = (float *)malloc(kNumBytes);
	float *h_B_ = (float *)malloc(kNumBytes);
	float *h_C_ = (float *)malloc(kNumBytes);

	if (h_A_ == NULL || h_B_ == NULL || h_C_ == NULL) {
		std::cerr << "La memoria ha fallado lol \n";
		getchar();
		exit(-1);
	}
	float* d_A_ = NULL;
	float* d_B_ = NULL;
	float* d_C_ = NULL;

	//declaro los vectores en la GPU (HOST)
	hipMalloc((void **)&d_A_, kNumBytes);
	hipMalloc((void **)&d_B_, kNumBytes);
	hipMalloc((void **)&d_C_, kNumBytes);

	//le ponemos numeros aleatorios en lugar de los que les apetezca al cacharro

	for (int i = 0; i < kNumElements; ++i)
	{
		h_A_[i] = rand() / RAND_MAX;
		h_B_[i] = rand() / RAND_MAX;
 	}
	
	//PASO 3: Transferencia CPU a la GPU

	hipMemcpy(d_A_, h_A_, kNumBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B_, h_B_, kNumBytes, hipMemcpyHostToDevice);

	//PASO 4: Ejecución de le kernel

	const int threads_per_block_ = 256;
	const int blocks_per_grid_ = 650000;

	dim3 block(threads_per_block_, 1, 1);
	dim3 grid(blocks_per_grid_, 1, 1);

	suma_vectores_grande<<<grid, block >>>(d_A_, d_B_, d_C_, kNumElements);
	hipError_t err_ = hipGetLastError();
	if (err_ != hipSuccess)
	{
		std::cerr << hipGetErrorString(err_) << "\n";
		getchar();
		exit(-1);
	}

	//paso 5: transferencia de la gpu a la cpu

	hipMemcpy(h_C_, d_C_, kNumBytes, hipMemcpyDeviceToHost);

	//paso 6: Comprobación y liberación

	for (int i = 0; i < kNumElements; i++) {
		if (fabs(h_A_[i] + h_B_[i] - h_C_[i]) > 1e-5) {
			std::cerr << "fallo de verificación en la posicion" << i << "\n";
		}
	}

	free(h_A_);
	free(h_B_);
	free(h_C_);
	hipFree(d_A_);
	hipFree(d_B_);
	hipFree(d_C_);

	hipDeviceReset();
	std::cout << "Test Passed \n";
	getchar();
}
